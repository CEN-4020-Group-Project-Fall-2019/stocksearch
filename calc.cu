#include "hip/hip_runtime.h"
#include "calc.h"

#include <hip/hip_cooperative_groups.h>
using namespace cooperative_groups;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void movingAvg(int n, int numDays, double* in, double* out){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	if(i<n){
		out[i] = 0;
		
		if(i>=numDays-1){
			for(int j = i-numDays+1; j <= i; j++){
				out[i] += in[j];
			}
			out[i] /= numDays;
		}
	}
}

__global__ void deltas(int n, double* in, double* out){
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	
	if(i<n){
		if(i==0) out[i] = 0;
		else{
			out[i] = in[i] - in[i-1];
		}
	}
}

__global__ void stdDev(int n, int period, double* x, double* std){
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	
	if(i <= period){
		std[i] = 0;
	}
	
	if(i < n && i > period){
		double average = 0;
		for(int j = i-period+1; j <= i; j++){
			average += x[j];
		}
		
		average /= period;
		double deviation = 0;
		
		for(int j = i-period+1; j <= i; j++){
			deviation += (x[j] - average) * (x[j] - average);
		}
		
		deviation /= period;
		std[i] = __dsqrt_rn(deviation);
	}
}

__global__ void normalize(int* n, double* ave, double* in){
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	
	if(i == *n){
		*ave = 0;
		for(int j = 0; j < *n; j++){
			(*ave) = (*ave) + in[j];
		}
		(*ave) = (*ave) / *n;
	}
	
	__syncthreads();
	
	if(i < *n){
		in[i] /= (*ave);
	}
}

__global__ void pearson(int nx, double* x, int* xDates, int ny, double* y, int* yDates, double* r, double* num, double* Xden, double* Yden, double* aveX, double* aveY){
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	auto g = this_grid();

	if(i==nx){
		double sumX = 0, sumY = 0;
		for(int j = 0; j < nx; j++){
			sumX += x[j];
		}
		for(int j = 0; j < ny; j++){
			sumY += y[j];
		}
		*aveX = sumX/nx;
		*aveY = sumY/ny;
	}

	g.sync();
	if(i<nx){
		int j;
		for(j = 0; j < ny; j++){
			if(xDates[i] > yDates[j] && xDates[i] < yDates[j+1]){
				break;
			}
		}
		num[i] = (x[i]-*aveX)*(y[j]-*aveY);
		Xden[i] = ((x[i]-*aveX)*(x[i]-*aveX));
		Yden[i] = ((y[j]-*aveY)*(y[j]-*aveY));
	}

	g.sync();

	if(i==nx){
		double numerator = 0, d1 = 0, d2 = 0;
		for(int j = 0; j < nx; j++){
			numerator += num[j];
			d1 += Xden[j];
			d2 += Yden[j];
		}
		*r = numerator/((__dsqrt_rn(d1))*(__dsqrt_rn(d2)));
	}
}


double* calcPearson(int nx, double* x, int* xDates, int ny, double* y, int* yDates){
	double* correlation = new double;
	double* d_r, *d_aveX, *d_aveY;

	hipMalloc((void**)&d_r, sizeof(double));
	hipMalloc((void**)&d_aveX, sizeof(double));
	hipMalloc((void**)&d_aveY, sizeof(double));

	double* d_numerator, *d_Xden, *d_Yden;

	if(nx > ny){
		gpuErrchk(hipMalloc((void**)&d_numerator, nx*sizeof(double)));
		gpuErrchk(hipMalloc((void**)&d_Xden, nx*sizeof(double)));
		gpuErrchk(hipMalloc((void**)&d_Yden, nx*sizeof(double)));
		pearson<<<(nx)/1024 + 1, 1024>>>(nx, x, xDates, ny, y, yDates, d_r, d_numerator, d_Xden, d_Yden, d_aveX, d_aveY);
	}
	else{
		gpuErrchk(hipMalloc((void**)&d_numerator, ny*sizeof(double)));
		gpuErrchk(hipMalloc((void**)&d_Xden, ny*sizeof(double)));
		gpuErrchk(hipMalloc((void**)&d_Yden, ny*sizeof(double)));
		pearson<<<(ny)/1024 + 1, 1024>>>(ny, y, yDates, nx, x, xDates, d_r, d_numerator, d_Xden, d_Yden, d_aveX, d_aveY);
	}

	hipMemcpy(correlation, d_r, sizeof(double), hipMemcpyDeviceToHost);

	hipFree(d_numerator);
	hipFree(d_Xden);
	hipFree(d_Yden);
	hipFree(d_r);
	hipFree(d_aveX);
	hipFree(d_aveY);

	return correlation;
}

int parseDate(char* date){
	char* year, *month, *day;
	bool first = false;
	year = date;
	
	for(int i = 0; date[i] != '\0'; i++){
		if(date[i] == '-'){
			date[i] = '\0';
			if(!first){
				first = true;
				month = &date[i+1];
			}
			else{
				day = &date[i+1];
			}
		}
	}
	return stoi(year)*365 + stoi(month)*30 + stoi(day);
}

__global__ void optionPrice(double* stockPrices, int numDays, double* strikes, double* maturity, bool* call, double* optionPrices, int numOptions){
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	double R = 1.0202;
	__shared__ double deviation;
	
	if(threadIdx.x == 0){
		double average = 0;
		
		for(int j = 0; j <= numDays; j++){
			average += stockPrices[j];
		}
		average /= numDays;
		deviation = 0;
		for(int j = 0; j <= numDays; j++){
			double in = (stockPrices[j] - average) * (stockPrices[j] - average);
			deviation += in;
		}
		deviation /= numDays-1;
		deviation = __dsqrt_rn(deviation);
		deviation *= __dsqrt_rn(numDays);
		deviation /= average;
	}

	__syncthreads();
	if (i < numOptions){
		if(call[i]){
			double priceUp = stockPrices[numDays-1] + stockPrices[numDays-1] * deviation * maturity[i];
			double priceDown = stockPrices[numDays-1] - stockPrices[numDays-1] * deviation * maturity[i];
			if(priceDown < 0) priceDown = 0;
			double maxUp = priceUp - strikes[i];
			if(maxUp < 0) maxUp = 0;
			double maxDown = priceDown - strikes[i];
			if(maxDown < 0) maxDown = 0;
			double valPrice = (maxUp-maxDown)/(priceUp-priceDown);
			optionPrices[i] = (stockPrices[numDays-1] * valPrice) + ((maxUp - (priceUp * valPrice))*exp(R*maturity[i]));
		}
	}

	
}
__global__ void launch(double** prices, int* sizes, int n, int* status, double** optionPrices, double** strikes, double** exp, bool** calls, int* numOptions){ //rewrite to avoid calling kernels from kernels
	int i = blockIdx.x*blockDim.x+threadIdx.x; //there might be an issue with launching like this for single or cmdline symbols as this might restrict number of threads launched, I really don't know what the limitations of it are
	
	if(i < n){
		printf("in kernel %d\n", sizes[i]);
		if(sizes[i] > 10){
			unsigned int blocks = 512/sizes[i]+1, threads = 512;
			double* ten = new double[sizes[i]], *five = new double[sizes[i]];
			double* d1, *d5, *d10, *d2_5, *d2_10, *std;
			
			d1 = new double[sizes[i]];
			d5 = new double[sizes[i]];
			d10 = new double[sizes[i]];
			d2_5 = new double[sizes[i]];
			d2_10 = new double[sizes[i]];
			std = new double[sizes[i]];
			double* ave = new double;
			hipDeviceSynchronize();

			movingAvg<<<blocks, threads>>>(sizes[i], 10, prices[i], five); //calc moving average over 10 days for entire data present for this thread's symbol
			hipDeviceSynchronize();
			movingAvg<<<blocks, threads>>>(sizes[i], 20, prices[i], ten); //20 day mvg avg
			hipDeviceSynchronize();
			deltas<<<blocks, threads>>>(sizes[i], prices[i], d1); //differences between every day of data for symbol
			hipDeviceSynchronize();
			deltas<<<blocks, threads>>>(sizes[i], five, d5);	//5 day differences, which is really ten day because of setting above. I never changed the array name
			hipDeviceSynchronize();
			deltas<<<blocks, threads>>>(sizes[i], ten, d10);	//20 day difference
			hipDeviceSynchronize();
			deltas<<<blocks, threads>>>(sizes[i], d5, d2_5);	//d^2 of 10 day, difference of 10 day differences
			hipDeviceSynchronize();
			deltas<<<blocks, threads>>>(sizes[i], d10, d2_10);	//d^2 of 20 day
			hipDeviceSynchronize();
			stdDev<<<blocks, threads>>>(sizes[i], 20, prices[i], std); //standard deviation of previous 20 days for every day in existence (rolling std avg of symbol data)
			hipDeviceSynchronize();
			
			if(sizes[i] >= 253){//if more than a year of data, calculate options data given volatility and prices of options
				optionPrice<<<numOptions[i]/512+1, 512>>>(&(prices[i][sizes[i]-253]), 252, strikes[i], exp[i], calls[i], optionPrices[i], numOptions[i]);
			}			
			else{
				optionPrice<<<numOptions[i]/512+1, 512>>>(prices[i], sizes[i], strikes[i], exp[i], calls[i], optionPrices[i], numOptions[i]);
			}
			hipDeviceSynchronize();

			normalize<<<blocks, threads>>>(&sizes[i], ave, std);
			status[i] = 0;
			int index = sizes[i]-1;
			
			if((d10[index] < 0.02 || d10[index] > -0.02) && d2_5[index] > 0 && d5[index] > 0 && std[index] < 2){//if the change in 20 day around 0 and change in 10 day change > 0 and stdev isn't some psychotic amount
				status[i] = 1;	//go long!
			}
			else if( ((d2_5[index] < 0 && (zero(d5[index]) || d5[index] < -0.002)) ) || (d2_10[index] < 0 && (zero(d10[index]) || d10[index] < -.002)) ) {
				status[i] = 2;	//short that
			}
			else if(prices[i][index-1] < five[index-1] && prices[i][index] > five[index] && std[index] > 1){
				status[i] = 2;	//short that
			}
			hipDeviceSynchronize();

			delete[] ten;
			delete[] five;
			delete[] d1;
			delete[] d5;
			delete[] d10;
			delete[] d2_5;
			delete[] d2_10;
			delete[] std;
			delete ave;
		}
	}
}

int main(int argc, char** argv) {
	gpuErrchk(hipSetDevice(1));
	hipDeviceSynchronize();

	vector<char*>* symbols = new vector<char*>();

	if(argc <= 1)
		listDir("./proc", symbols);
	else{
		for (int i = 1; i < argc; i++){
			char* temp = new char[strlen(argv[i])+5];
			strcpy(temp, argv[i]);
			cout<<"Reading "<<temp<<endl;
			addStrToVec(strcat(temp,".csv"), symbols);
			delete [] temp;
		}

	}

	vector<AVData*> dataList = vector<AVData*>();
	vector<OptionData*> optionList = vector<OptionData*>();

	for(int i = 0; i < symbols->size(); i++){
		char* dirName = new char[100];
		strcpy(dirName, "./proc/");
		AVData* temp = new AVData(strcat(dirName, (*symbols)[i]));
		dataList.push_back(temp);

		strcpy(dirName, "./options/");
		OptionData* temp2 = new OptionData(strcat(dirName, (*symbols)[i]));
		optionList.push_back(temp2);
	}

	double** d_prices, **prices = new double*[dataList.size()];
	int* d_pSizes, *pSizes = new int[dataList.size()];

	gpuErrchk(hipMalloc((void**)&d_pSizes, dataList.size()*sizeof(int)));
	gpuErrchk(hipMalloc((void**)&d_prices, dataList.size()*sizeof(double*)));
	hipDeviceSynchronize();
	size_t totalSize = 0;
	for(int i = 0; i < dataList.size(); i++){
		
		dataList[i]->tokenize();
		size_t size = dataList[i]->price()->size()*sizeof(double);
		gpuErrchk(hipMalloc((void**)&(prices[i]), size));
		hipDeviceSynchronize();
		gpuErrchk(hipMemcpy(prices[i], dataList[i]->price()->data(), size, hipMemcpyHostToDevice));
		hipDeviceSynchronize();
		pSizes[i] = dataList[i]->price()->size();
		totalSize += dataList[i]->price()->size()*sizeof(double)*9 + sizeof(double);
	}
	
	hipDeviceSynchronize();
	gpuErrchk(hipMemcpy(d_pSizes, pSizes, dataList.size()*sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_prices, prices, dataList.size()*sizeof(double), hipMemcpyHostToDevice));
	hipDeviceSynchronize();

	int * d_status, *status = new int[dataList.size()];
	hipMalloc((void**)&d_status, dataList.size()*sizeof(int));

	hipDeviceSynchronize();
	size_t cudaMem, cudaMemTotal;//, wrapperSize = dataList.size()*sizeof(int);
	gpuErrchk(hipMemGetInfo(&cudaMem, &cudaMemTotal));
	cout<<"Allocating "<<totalSize<<" bytes on device\n";

	gpuErrchk(hipDeviceSetLimit(hipLimitMallocHeapSize, totalSize));

	double **d_optionPrices = 0, **d_strikes = 0, ** optionPrices= new double*[optionList.size()], **strikes = new double*[optionList.size()];
	int *numOptions = 0, *d_numOptions = 0;
	double **d_exp = 0, **exp = new double*[optionList.size()];
	bool** d_call = 0, **call = new bool*[optionList.size()];

	numOptions = new int[optionList.size()];
	hipMalloc((void**)&d_optionPrices, optionList.size()*sizeof(double*));
	hipMalloc((void**)&d_exp, optionList.size()*sizeof(double*));
	hipMalloc((void**)&d_call, optionList.size()*sizeof(bool*));
	hipMalloc((void**)&d_strikes, optionList.size()*sizeof(double*));
	hipMalloc((void**)&d_numOptions, optionList.size()*sizeof(int));

	for(int i = 0; i < optionList.size(); i++){
		optionList[i]->tokenize();
		hipMalloc((void**)&(optionPrices[i]), optionList[i]->call.size()*sizeof(double));
		hipMalloc((void**)&(exp[i]), optionList[i]->exp.size()*sizeof(double));
		hipMalloc((void**)&(call[i]), optionList[i]->call.size()*sizeof(bool));
		hipMalloc((void**)&(strikes[i]), optionList[i]->strike.size()*sizeof(double));

		hipMemcpy(exp[i], optionList[i]->exp.data(), optionList[i]->exp.size()*sizeof(double), hipMemcpyHostToDevice);
		hipMemcpy(call[i], optionList[i]->call.data(), optionList[i]->call.size()*sizeof(bool), hipMemcpyHostToDevice);
		hipMemcpy(strikes[i], optionList[i]->strike.data(), optionList[i]->strike.size()*sizeof(double), hipMemcpyHostToDevice);
		numOptions[i] = optionList[i]->call.size();
	}

	gpuErrchk( hipMemcpy(d_exp, exp, optionList.size()*sizeof(double*), hipMemcpyHostToDevice));
	gpuErrchk( hipMemcpy(d_optionPrices, optionPrices, optionList.size()*sizeof(double*), hipMemcpyHostToDevice));
	hipMemcpy(d_call, call, optionList.size()*sizeof(bool*), hipMemcpyHostToDevice);
	hipMemcpy(d_strikes, strikes, optionList.size()*sizeof(double*), hipMemcpyHostToDevice);
	hipMemcpy(d_numOptions, numOptions, optionList.size()*sizeof(int), hipMemcpyHostToDevice);

	hipDeviceSynchronize();
	printf("launching kernel\n");
	launch<<<dataList.size()/512+1, 512>>>(d_prices, d_pSizes, dataList.size(), d_status, d_optionPrices, d_strikes, d_exp, d_call, d_numOptions);
	hipDeviceSynchronize();

	for(int i = 0; i < optionList.size(); i++){
		double *temp = new double[optionList[i]->call.size()];
		hipMemcpy(temp, optionPrices[i], optionList[i]->call.size()*sizeof(double), hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		hipFree(optionPrices[i]);

		optionPrices[i] = temp;
		optionList[i]->comparePrices(optionPrices[i]);

		hipFree(exp[i]);
		hipFree(strikes[i]);
		hipFree(call[i]);
	}

	hipFree(d_exp);
	hipFree(d_strikes);
	hipFree(d_call);
	hipFree(d_numOptions);
	hipFree(d_optionPrices);
	hipDeviceSynchronize();

	hipMemcpy(status, d_status, dataList.size()*sizeof(int), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	cout<<"-------------------Long-------------------\n";
	
	for(int i = 0; i < dataList.size(); i++){
		if(status[i] == 1)
			cout<<dataList[i]->fileName()<<endl;
	}
	cout<<"-------------------Short-------------------\n";
	
	for(int i = 0; i < dataList.size(); i++){
		if(status[i] == 2)
			cout<<dataList[i]->fileName()<<endl;
		hipFree(prices[i]);
	}

	hipFree(d_pSizes);
	hipFree(d_prices);

	delete[] optionPrices;
	delete[] exp;
	delete[] strikes;
	delete[] call;
	delete[] prices;
	delete[] pSizes;
	delete[] status;

	return 0;
}

__device__ bool zero(double in){
	double thresh = .002;
	if(in < thresh && in > -thresh){
		return true;
	}
	return false;
}

CSVReader::CSVReader(char* fn){
	setFile(fn);
}

void CSVReader::setFile(char* fn){
	file.open(fn);
	filename = fn;//memory leak, causes segfault if initialized from argv. Need to implement copy iterator for strings.
}

vector<char*>* CSVReader::getline(){
	vector<char*>* tokenized;
	if(!this->eof()){
		tokenized = new vector<char*>;//mem leak?
		if(!file.eof()){
			char* line = new char[256];
			file.getline(line, 256);
			tokenized->push_back(&line[0]);
			for(int i = 0; line[i] != '\0'; i++){
				if(line[i]==','){
					line[i] = '\0';
					tokenized->push_back(&line[i+1]);
				}
			}	
		}
	}
	return tokenized;	
}

bool CSVReader::eof(){
	return file.eof();
}

bool CSVReader::isOpen(){
	return file.is_open();
}

CSVReader::~CSVReader(){//eventually add delete[] filename;
	file.close();
}

CSVReader::CSVReader(const CSVReader& copy){
	filename = copy.filename;
	file.open(filename);
}

const char* CSVReader::fileName(){
	return filename;
}

void AVData::tokenize(){
	if(isOpen()){
		while(vector<char*>* tokenized = this->getline()){
			if(!this->eof()){
				if(strcmp((*tokenized)[1], "") != 0)
					open.push_back(stod((*tokenized)[1]));
				if(strcmp((*tokenized)[2], "") != 0)
					high.push_back(stod((*tokenized)[2]));
				if(strcmp((*tokenized)[3], "") != 0)
					low.push_back(stod((*tokenized)[3]));
				if(strcmp((*tokenized)[4], "") != 0)
					close.push_back(stod((*tokenized)[4]));
				if(strcmp((*tokenized)[5], "") != 0)
					volume.push_back(stod((*tokenized)[5]));
			}
			else break;
		}
	}
}

AVData::AVData(char* fn) : CSVReader(fn){}

void addStrToVec(char* str, vector<char*>* v){
	char* temp = new char[strlen(str)+1];
	memcpy(temp, str, strlen(str)+1);
	v->push_back(temp);
}

void listDir(char* dirPath, vector<char*>* v){
	DIR* dirp = opendir(dirPath);
	struct dirent * dp;
	for(int i = 0; (dp = readdir(dirp)) != NULL; i++){
		if(i >= 2) addStrToVec(dp->d_name, v);
	}
	closedir(dirp);
}

const vector<double>* AVData::price(){
	return &close;
}

OptionData::OptionData(char* fn) : CSVReader(fn){}

void OptionData::tokenize(){
	if(isOpen()){
		time_t curTime = time(0);
		while(vector<char*>* tokenized = this->getline()){
			if(!this->eof()){

				call.push_back((strcmp((*tokenized)[0], "C") == 0));
				exp.push_back((stod((*tokenized)[1])+16*3600 - curTime)/(365*24*60*60));
				strike.push_back(stod((*tokenized)[2]));
				bid.push_back(stod((*tokenized)[3]));
				ask.push_back(stod((*tokenized)[4]));
				impVol.push_back(stod((*tokenized)[5]));

			}
			else break;
		}
	}
}

bool* OptionData::comparePrices(double* optionPrices){
	bool* temp = new bool[ask.size()];
	for(int i = 0; i < ask.size(); i++){
		temp[i] = optionPrices[i] < ask[i];
		if(call[i] && temp[i])
			printf("%s %f %f %f %f %f\n", this->fileName(), exp[i], strike[i], optionPrices[i], ask[i], bid[i]);
	}
	return temp;
}